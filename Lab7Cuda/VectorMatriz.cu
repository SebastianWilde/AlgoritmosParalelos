#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <stdio.h>

__global__ 
void VecMatkernel(float *d_A, float *d_B, float *d_C, int fil,int col)
{
	int IdX = threadIdx.x + blockIdx.x*blockDim.x;
	float sum = 0;
	if (IdX < fil)
	{
		for (int i = 0; i<col; i++)
			sum += d_B[i] * d_A[(i*col) + IdX];
		d_C[IdX] = sum;
	}
}

void LLenarMatrix(float*mat, int fil, int col)
{
	int i;
	for (i = 0; i < fil*col; i++)
			mat[i] = i * 2;
}

void Print(float*mat, int fil, int col)
{
	int i, j;
	for (i = 0; i < fil; i++)
	{
		for (j = 0; j < col; j++)
			printf("%3.0f ", mat[i*col + j]);
		printf("\n");
	}

}

int main()
{
	float *A, *B, *C, *d_A, *d_B, *d_C;
	int n_fila, n_columna;
	n_fila = n_columna = 5;
	A = (float*)malloc(sizeof(float)*n_fila*n_columna);
	B = (float*)malloc(sizeof(float)*n_columna);
	C = (float*)malloc(sizeof(float)*n_fila);
	LLenarMatrix(A,n_fila,n_columna);
	LLenarMatrix(B, n_columna, 1);
	printf("La matriz es\n");
	Print(A, n_fila, n_columna);
	printf("El vector es\n");
	Print(B, 1, n_columna);
	hipMalloc((void**)& d_A, sizeof(float)*n_fila*n_columna);
	hipMalloc((void**)& d_B, sizeof(float)*n_columna);
	hipMalloc((void**)& d_C, sizeof(float)*n_fila);
	hipMemcpy(d_A, A, sizeof(float)*n_fila*n_columna, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float)*n_columna, hipMemcpyHostToDevice);
	int N = n_fila*n_columna;
	VecMatkernel << < N/ 8 + 1, 8 >> >(d_A, d_B, d_C, n_fila,n_columna);
	hipMemcpy(C, d_C, sizeof(float)*n_fila, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	printf("La rpta es\n");
	Print(C, n_fila, 1);
	free(A);
	free(B);
	free(C);
	system("PAUSE");
    return 0;
}