#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void vecMulKernel(float* a, float* b, float* c, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n) c[i] = a[i] * b[0];
}

int main(void) 
{
	int N = 24;
	int i;
	float *A, *B, *C, *d_A, *d_B, *d_C;
	int size = N * sizeof(float);
	A=(float*)malloc(size);
	B=(float*)malloc(sizeof(float));
	C=(float*)malloc(size);
	//inicializacion de los vectores
	for (i=0;i<N;i++)
	{
		A[i]=i;
	}
	B[0]=10;
	//impresion vectores
	printf("Primer vector\n");
	for (i=0;i<4;i++)
		printf ("%4.1f ", A[i] );
	printf("\n");

	//Memoria en GPU
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, sizeof(float));
	hipMalloc((void **) &d_C, size);
	//Copiar datos
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float), hipMemcpyHostToDevice);
	//Calculo de bloques
	int block_size=8;
	int n_blocks=N/block_size + (N%block_size == 0 ? 0:1);
	// Kernel invocation 
	vecMulKernel <<< n_blocks, block_size >>> (d_A, d_B, d_C, N);
	//vecAddKernel<<<ceil(N/8.0), 8>>>(d_A, d_B, d_C, N);	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	printf("Respuesta\n");
	for (i=0;i<4;i++)
		printf ("%4.1f ", C[i] );
	printf("\n");
	free(A);
	free(B);
	free(C);
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree (d_C);	
	return 0;
}