#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void vecAddKernel(float* a, float* b, float* c, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n) c[i] = a[i] + b[i];
}

int main(void) 
{
	int N = 24;
	int i;
	float *A, *B, *C, *d_A, *d_B, *d_C;
	int size = N * sizeof(float);
	A=(float*)malloc(size);
	B=(float*)malloc(size);
	C=(float*)malloc(size);
	//inicializacion de los vectores
	for (i=0;i<N;i++)
	{
		A[i]=i;
		B[i]=i*i-1;
	}
	//impresion vectores
	printf("Primer vector\n");
	for (i=0;i<4;i++)
		printf ("%4.1f ", A[i] );
	printf("\n");
	printf("Segundo vector\n");
	for (i=0;i<4;i++)
		printf ("%4.1f ", B[i] );
	printf("\n");

	//Memoria en GPU
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_C, size);
	//Copiar datos
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	//Calculo de bloques
	int block_size=8;
	int n_blocks=N/block_size + (N%block_size == 0 ? 0:1);
	// Kernel invocation 
	vecAddKernel <<< n_blocks, block_size >>> (d_A, d_B, d_C, N);
	//vecAddKernel<<<ceil(N/8.0), 8>>>(d_A, d_B, d_C, N);	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	printf("Respuesta\n");
	for (i=0;i<4;i++)
		printf ("%4.1f ", C[i] );
	printf("\n");
	free(A);
	free(B);
	free(C);
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree (d_C);	
	return 0;
}