#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>

__global__
void vecMulKernel(float* a, int b, float* c, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<n) c[i] = a[i] * b;
}

int main(void)
{
	int N = 24;
	int i;
	int B = 10;
	float *A,*C, *d_A,*d_C;
	int size = N * sizeof(float);
	A = (float*)malloc(size);
	C = (float*)malloc(size);
	//inicializacion de los vectores
	for (i = 0; i<N; i++)
	{
		A[i] = i;
	}
	//impresion vectores
	printf("Primer vector\n");
	for (i = 0; i<4; i++)
		printf("%4.1f ", A[i]);
	printf("\n");
	//Memoria en GPU
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_C, size);
	//Copiar datos
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	//Calculo de bloques
	int block_size = 8;
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);
	// Kernel invocation 
	vecMulKernel <<< n_blocks, block_size >>> (d_A, B, d_C, N);
	//vecAddKernel<<<ceil(N/8.0), 8>>>(d_A, d_B, d_C, N);	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	printf("Respuesta\n");
	for (i = 0; i<4; i++)
		printf("%4.1f ", C[i]);
	printf("\n");
	free(A);
	free(C);
	hipFree(d_A);
	hipFree(d_C);
	system("PAUSE");
	return 0;
}