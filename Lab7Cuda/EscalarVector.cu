#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>

__global__
void vecMulKernel(float* a, int b, float* c, int nx, int ny)
{
	int IdX = threadIdx.x + blockDim.x * blockIdx.x; //Columna
	int IdY = blockIdx.y*blockDim.y + threadIdx.y; //Filas
	if (IdX<nx && IdY<ny) c[IdY*nx + IdX] = a[IdY*nx + IdX] * b;
}

int main(void)
{
	int n_columnas = 24;
	int n_filas = 24;
	int i,j;
	int B = 10;
	int N = n_columnas*n_filas;
	float *A,*C, *d_A,*d_C;
	int size = N * sizeof(float);
	A = (float*)malloc(size);
	C = (float*)malloc(size);
	//inicializacion de los vectores
	for (i = 0; i<N; i++)
	{
		A[i] = i;
	}
	//impresion vectores
	printf("Primer vector\n");
	for (i = 0; i < 4; i++)
	{
		for (j = 0; j < 4;j++)
			printf("%4.1f ", A[i*n_columnas+j]);
		printf("\n");
	}

	//Memoria en GPU
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_C, size);
	//Copiar datos
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	/*
	//Calculo de bloques
	int block_size = 8;
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);
	*/
	dim3 DimGrid((n_columnas - 1) / 16 + 1, (n_filas - 1) / 16 + 1, 1);
	dim3 DimBlock(16, 16, 1);
	// Kernel invocation 
	vecMulKernel <<< DimGrid, DimBlock >>> (d_A, B, d_C, n_columnas,n_filas);
	//vecAddKernel<<<ceil(N/8.0), 8>>>(d_A, d_B, d_C, N);	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// Free device memory for A, B, C
	printf("Respuesta\n");
	for (i = 0; i < 4; i++)
	{
		for (j = 0; j < 4; j++)
			printf("%4.1f ", C[i*n_columnas + j]);
		printf("\n");
	}
	free(A);
	free(C);
	hipFree(d_A);
	hipFree(d_C);
	system("PAUSE");
	return 0;
}