#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <stdio.h>


__global__ 
void foo_kernel(float *R,float *S, const int nx, const int ny, const int nz)
{
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y* blockDim.y + threadIdx.y;
	unsigned int zIndex = blockIdx.z* blockDim.z + threadIdx.z;

	if ((xIndex < nx) && (yIndex < xIndex) && (zIndex < nz))
	{
		unsigned int index = xIndex + nx*yIndex + nx*ny*zIndex;
		S[index] = R[index]*2;
	}
}

int main()
{
	//Variables espacio host
	float *A, *B,*d_A,*d_B;
	int nx, ny, nz,i;
	nx = ny = nz = 16;
	int size = nx*ny*nz*sizeof(float);
	A = (float*)malloc(size);
	B = (float*)malloc(size);
	for (i = 0; i<nx*ny*nz; i++)
	{
		A[i] = i%10;
	}
	//Varables device
	hipMalloc((void**)&d_A,size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_B, size);
	//Calculo de bloques
	int threadsInX = 8;
	int threadsInY = 8;
	int threadsInZ = 8;
	int blocksInX = (nx + threadsInX - 1) / threadsInX;
	int blocksInY = (ny + threadsInY - 1) / threadsInY;
	int blocksInZ = (nz + threadsInZ - 1) / threadsInZ;
	dim3 Dg = dim3(blocksInX, blocksInY*blocksInZ);
	dim3 Db = dim3(threadsInX, threadsInY, threadsInZ);
	foo_kernel <<<Dg,Db>>>(A,B, nx, ny, nz);
	system("PAUSE");
	return 0;
}
